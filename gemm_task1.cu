#include <iostream>
#include <iomanip>
#include "mkl.h"
#include "hipblas.h"
#include "utils.h"

#define SINGLE_PRECISION //Comment out to use double precision arithmetic
#define DOUBLE_PRECISION

#ifdef SINGLE_PRECISION
	#define elem_t float
	#define blasGemm cblas_sgemm 
	#define cublasGemm hipblasSgemm
	#define cublasGemmBatched hipblasSgemmBatched
#elif defined(DOUBLE_PRECISION)
	#define elem_t double
	#define blasGemm cblas_dgemm 
	#define cublasGemm hipblasDgemm
	#define cublasGemmBatched hipblasDgemmBatched
#endif

#ifndef GEMM_M
#define GEMM_M 256
#endif
#ifndef GEMM_N
#define GEMM_N 256
#endif
#ifndef GEMM_K
#define GEMM_K 256
#endif

#ifndef TILE_M
#define TILE_M 64
#endif
#ifndef TILE_N
#define TILE_N 64
#endif

#ifndef NB_STREAMS
#define NB_STREAMS 16
#endif

#ifndef WARMUPS
#define WARMUPS 1
#endif
#ifndef ITERS
#define ITERS 10
#endif

void tileGemm(hipblasHandle_t handle, int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC, int tileM, int tileN)
{
	//TODO: TASK 3
}

void tileGemmStreams(hipblasHandle_t handle, int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC, int tileM, int tileN, int nb_streams, hipStream_t *streams)
{
	//TODO: TASK 4
}

void tileGemmBatch(hipblasHandle_t handle, int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC, int tileM, int tileN)
{
	//TODO: TASK 5
}

int main(int argc, char **argv)
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipStream_t *streams;
       	createStreams(NB_STREAMS, &streams);

	float *times = new float[2*ITERS];
	float *timesCPU = times;
	float *timesGPU = times + ITERS;

	elem_t *A, *B, *C, *Cgpu;
	elem_t *d_A, *d_B, *d_C;
	int M = GEMM_M;
	int N = GEMM_N;
	int K = GEMM_K;

	//TASK 1 (Allocate and init A,B,C)
	allocateMatrixCPU(M,K,&A);
	allocateMatrixCPU(K,N,&B);
	allocateMatrixCPU(M,N,&C);

	initMatrixRandomCPU<elem_t>(M,K,A);
	initMatrixRandomCPU<elem_t>(K,N,B);
	initMatrixCPU<elem_t>(M,N,C,0.0);

	//TODO: TASK 2.1 (Allocate and init d_A, d_B, d_C)

	elem_t alpha = 1.0;
	elem_t beta = 0.0;

	//CPU
	struct timespec cpu_start, cpu_end;
	for (int i=0; i<ITERS; i++)
	{
		clock_gettime(CLOCK_MONOTONIC, &cpu_start);
		blasGemm(CblasColMajor, CblasNoTrans, CblasNoTrans, M, N, K, alpha, A, M, B, K, beta, C, M);
		clock_gettime(CLOCK_MONOTONIC, &cpu_end);
		timesCPU[i] = computeCPUTime(&cpu_start, &cpu_end);
	}

	//GPU
	for (int i=0; i<WARMUPS; i++)
	{
		//TODO: TASK 2.2 (run cublasGemm)
		//TODO: TASK 3
		//TODO: TASK 4
		//TODO: TASK 5
		hipDeviceSynchronize();
	}
	hipEvent_t gpu_start, gpu_end;
	for (int i=0; i<ITERS; i++)
	{
		//TODO: TASK 2.2 (run cublasGemm)
		//TODO: TASK 3
		//TODO: TASK 4
		//TODO: TASK 5
		//TODO: TASK 2.2 (Measure execution times)
		hipDeviceSynchronize();
	}

	//TASK 1 (Compute and print average execution time/performance on CPU)
	float flops = 2*(float)M*(float)N*(float)K;

	float avg_cpu=0.0;
	for (int i=0; i<ITERS; i++)
		avg_cpu += timesCPU[i];
	avg_cpu = avg_cpu / (float)ITERS;
	std::cout << "==== CPU ====\n";
	std::cout << "Execution time: " << avg_cpu << " ms.\n";
	std::cout << "Performance: " << (flops/1.0e9)/(avg_cpu/1.0e3) << " GFLOP/s.\n";

	//TODO: TASK 2.2 (Compute and print average execution time/performance on GPU)

	//TODO: TASK 2.2 (Compare CPU and GPU output)

	//TODO: TASK 2.1 (Free d_A, d_B, d_C)

	//TASK 1 (Free A,B,C)
	freeMatrixCPU(M,K,A);
	freeMatrixCPU(K,N,B);
	freeMatrixCPU(M,N,C);

	destroyStreams(NB_STREAMS, streams);
	hipblasDestroy(handle);

	delete[] times;

}
